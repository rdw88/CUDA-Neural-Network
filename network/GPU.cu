#include "GPU.h"

#include <iostream>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>


#pragma comment(lib, "cublas.lib")


static hipblasHandle_t cublasContext;



void createCublasContext() {
	hipblasCreate(&cublasContext);
}



void destroyCublasContext() {
	hipblasDestroy(cublasContext);
}



float *gpu_loadVector(std::vector<float> *vector) {
	unsigned int size = vector->size();
	float *pointer;

	hipMalloc(&pointer, size * sizeof(float));
	hipMemcpy(pointer, vector->data(), size * sizeof(float), hipMemcpyHostToDevice);
	
	return pointer;
}



float *gpu_loadVectorFromPointer(float *cpuPointer, size_t numFloats) {
	float *gpuPointer;

	hipError_t error = hipMalloc(&gpuPointer, numFloats * sizeof(float));
	error = hipMemcpy(gpuPointer, cpuPointer, numFloats * sizeof(float), hipMemcpyHostToDevice);

	return gpuPointer;
}



float *gpu_unloadVector(float *gpuPointer, size_t numFloats) {
	float *cpuPointer = (float *) malloc(numFloats * sizeof(float));

	hipMemcpy(cpuPointer, gpuPointer, numFloats * sizeof(float), hipMemcpyDeviceToHost);

	return cpuPointer;
}



void gpu_freeMemory(float *gpuPointer) {
	hipFree(gpuPointer);
}



void gpu_batchVectorMatrixMultiply(std::vector<float *> *matrices, std::vector<float *> *vectors, std::vector<float *> *results, int numColumns, int numRows, int batches) {
	float **gpuMatrixPointers;
	float **gpuVectorPointers;
	float **gpuResultPointers;

	hipMalloc(&gpuMatrixPointers, matrices->size() * sizeof(float *));
	hipMalloc(&gpuVectorPointers, vectors->size() * sizeof(float *));
	hipMalloc(&gpuResultPointers, results->size() * sizeof(float *));

	hipMemcpy(gpuMatrixPointers, &((* matrices)[0]), matrices->size() * sizeof(float *), hipMemcpyHostToDevice);
	hipMemcpy(gpuVectorPointers, &((* vectors)[0]), vectors->size() * sizeof(float *), hipMemcpyHostToDevice);
	hipMemcpy(gpuResultPointers, &((* results)[0]), results->size() * sizeof(float *), hipMemcpyHostToDevice);

	const float alpha = 1;
	const float beta = 1;
	const float *alphaRef = &alpha;
	const float *betaRef = &beta;

	int lda = numRows;
	int ldb = numColumns;
	int ldc = numRows;

	hipblasSgemmBatched(cublasContext, HIPBLAS_OP_N, HIPBLAS_OP_N, numRows, 1, numColumns, alphaRef, gpuMatrixPointers, lda, gpuVectorPointers, ldb, betaRef, gpuResultPointers, ldc, batches);

	hipFree(gpuMatrixPointers);
	hipFree(gpuVectorPointers);
	hipFree(gpuResultPointers);
}