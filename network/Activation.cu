#include "hip/hip_runtime.h"
#include "Activation.h"
#include <limits>


/**
 * Create a new Activation with the provided ActivationType.
 * 
 * @param activationType The ActivationType to use.
 * @return A new Activation.
 */
Activation newActivation(ActivationType activationType) {
    Activation activation;
    activation.activationType = activationType;
    activation.maxThreshold = std::numeric_limits<float>::max();
    activation.leakyReluGradient = 0;

    return activation;
}